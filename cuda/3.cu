#include "hip/hip_runtime.h"
#include<>
#include<hip/device_functions.h>
#include "utils.h"



__global__ void kernel_red_op(float * dop, const float * const dip, bool maxbool)
{
  
  extern __shared__ float dats[];

  int AID = threadIdx.x + blockDim.x * blockIdx.x;
  int thread_id = threadIdx.x;


  dats[thread_id] = dip[AID];
  __syncthreads();           
  for (unsigned int vars = blockDim.x / 2; vars > 0; vars >>= 1)
  {
    if (thread_id < vars)
    {
      if (maxbool)
        dats[thread_id] = max(dats[thread_id], dats[thread_id + vars]);
      else
        dats[thread_id] = min(dats[thread_id], dats[thread_id + vars]);
    }
    __syncthreads();        
  }


  if (thread_id == 0)
  {
    dop[blockIdx.x] = dats[0];
  }
}

__global__ void kern_hist(unsigned int * dop, const float * const dip,
  const size_t noofbins, float LLR, float minLL)
{
  int AID = threadIdx.x + blockDim.x * blockIdx.x;
  int b = (dip[AID] - minLL) / LLR * noofbins;
  if (b == noofbins)  b--;
  atomicAdd(&dop[b], 1);
}

__global__ void scan_kernel(unsigned int * dop, const float * const dip,
  const size_t noofbins, float LLR, float minLL)
{
  int AID = threadIdx.x + blockDim.x * blockIdx.x;
  int b = (dip[AID] - minLL) / LLR * noofbins;
  if (b == noofbins)  b--;
  atomicAdd(&dop[b], 1);
}


__global__ void cdfk(unsigned int * dip, const size_t noofbins)
{
  int AID = threadIdx.x;
  for (int d = 1; d < noofbins; d *= 2) {
    if ((AID + 1) % (d * 2) == 0) {
      dip[AID] += dip[AID - d];
    }
    __syncthreads();
  }
  if (AID == noofbins - 1) dip[AID] = 0;
  for (int d = noofbins / 2; d >= 1; d /= 2) {
    if ((AID + 1) % (d * 2) == 0) {
      unsigned int tmp = dip[AID - d];
      dip[AID - d] = dip[AID];
      dip[AID] += tmp;
    }
    __syncthreads();
  }
}


__global__ void cdfk_2(unsigned int * dip, const size_t noofbins)
{ 
  int idx = threadIdx.x;
  extern __shared__ int temp[];
  int outp = 0, inp = 1;

  temp[idx] = (idx > 0) ? dip[idx - 1] : 0;
  __syncthreads();

  for (int offs = 1; offs < n; offs *= 2) {
  
    outp = 1 - outp;
    inp = 1 - outp;
    if (idx >= offs) {
      temp[outp*n+idx] = temp[inp*n+idx - offs] + temp[inp*n+idx];  
    } else {
      temp[outp*n+idx] = temp[inp*n+idx];
    }
    __syncthreads();
  }
  dip[idx] = temp[outp*n+idx];
}

void myfuncs(const float* const d_LL,
  unsigned int* const d_cdf,
  float &minLL,
  float &max_LL,
  const size_t nr,
  const size_t nc,
  const size_t noofbins)
{

  const int varm = 1 << 10;
  int bl_grp = ceil((float)nc * nr / varm);

  float *middle_i;
  checkCudaErrors(hipMalloc(&middle_i, sizeof(float)* bl_grp)); 
  float *min_d, *max_d;
  checkCudaErrors(hipMalloc((void **)&min_d, sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&max_d, sizeof(float)));

  kernel_red_op << <bl_grp, varm, varm * sizeof(float) >> >(middle_i, d_LL, true);
  kernel_red_op << <1, bl_grp, bl_grp * sizeof(float) >> >(max_d, middle_i, true);
  kernel_red_op << <bl_grp, varm, varm * sizeof(float) >> >(middle_i, d_LL, false);
  kernel_red_op << <1, bl_grp, bl_grp * sizeof(float) >> >(min_d, middle_i, false);
  checkCudaErrors(hipMemcpy(&minLL, min_d, sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(&max_LL, max_d, sizeof(float), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(middle_i));
  checkCudaErrors(hipFree(min_d));
  checkCudaErrors(hipFree(max_d));


  float LLR = max_LL - minLL;
  printf("max_LL: %f  minLL: %f  LLR: %f\n", max_LL, minLL, LLR);

  checkCudaErrors(hipMemset(d_cdf, 0, sizeof(unsigned int)* noofbins));
  kern_hist << <bl_grp, varm >> >(d_cdf, d_LL, noofbins, LLR, minLL);


  cdfk_2 << <1, noofbins, sizeof(unsigned int) * noofbins * 2 >> >(d_cdf, noofbins);
}