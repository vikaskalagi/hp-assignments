#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"

__global__
void blur_g(const unsigned char* const ipc,
                   unsigned char* const opc,
                   int nr, int nc,
                   const float* const filt, const int fw)
{



  const int2 t2dp = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int t1dp = t2dp.y * nc + t2dp.x;

  if (t2dp.x >= nc || t2dp.y >= nr)
    return;

  float result = 0.f;
  
  for (int fr = -fw/2; fr <= fw/2; ++fr) {
    for (int fc = -fw/2; fc <= fw/2; ++fc) {

      int imr = min(max(t2dp.y + fr, 0), static_cast<int>(nr - 1));
      int imc = min(max(t2dp.x + fc, 0), static_cast<int>(nc - 1));

      float imval = static_cast<float>(ipc[imr * nc + imc]);
      float filval = filt[(fr + fw/2) * fw + fc + fw/2];

      result += imval * filval;
    }
  }

  opc[t1dp] = result;
}


__global__
void septc(const uchar4* const rgbaip,
                      int nr,
                      int nc,
                      unsigned char* const redc,
                      unsigned char* const greenc,
                      unsigned char* const bluec)
{

  const int2 t2dp = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int t1dp = t2dp.y * nc + t2dp.x;


  if (t2dp.x >= nc || t2dp.y >= nr)
    return;

  redc[t1dp] = rgbaip[t1dp].x;
  greenc[t1dp] = rgbaip[t1dp].y;
  bluec[t1dp] = rgbaip[t1dp].z;

}


__global__
void chanrecomb(const unsigned char* const redc,
                       const unsigned char* const greenc,
                       const unsigned char* const bluec,
                       uchar4* const rgbaop,
                       int nr,
                       int nc)
{
  const int2 t2dp = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int t1dp = t2dp.y * nc + t2dp.x;

  if (t2dp.x >= nc || t2dp.y >= nr)
    return;

  unsigned char red   = redc[t1dp];
  unsigned char green = greenc[t1dp];
  unsigned char blue  = bluec[t1dp];


  uchar4 oppix = make_uchar4(red, green, blue, 255);

  rgbaop[t1dp] = oppix;
}

unsigned char *dr, *dg, *db;
float         *dfilter;

void allocateMemoryAndCopyToGPU(const size_t nri, const size_t nci,
                                const float* const h_filter, const size_t fw)
{


  checkCudaErrors(hipMalloc(&dr,   sizeof(unsigned char) * nri * nci));
  checkCudaErrors(hipMalloc(&dg, sizeof(unsigned char) * nri * nci));
  checkCudaErrors(hipMalloc(&db,  sizeof(unsigned char) * nri * nci));


  int n_f_b = sizeof(float) * fw * fw;
  checkCudaErrors(hipMalloc(&dfilter, n_f_b));
umBytes, hipMemcpyHostToDevice);

  checkCudaErrors(hipMemcpy(dfilter, h_filter, n_f_b, hipMemcpyHostToDevice));

}

void your_gaussian_blur(const uchar4 * const hiprgb, uchar4 * const diprgb,
                        uchar4* const doprgb, const size_t nr, const size_t nc,
                        unsigned char *d_r_b, 
                        unsigned char *d_g_b, 
                        unsigned char *d_b_b,
                        const int fw)
{

  septc<<<gridSize, blockSize>>>(diprgb,nr,nc,dr,dg, db);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());


  blur_g<<<gridSize, blockSize>>>(dr,d_r_b,nr,nc, dfilter,fw);
  blur_g<<<gridSize, blockSize>>>(dg,d_g_b,nr,nc,dfilter,fw);
  blur_g<<<gridSize, blockSize>>>(db,d_b_b,nr,nc,dfilter,fw);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());


  chanrecomb<<<gridSize, blockSize>>>(d_r_b,d_g_b,d_b_b,doprgb,nr,nc);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}



void cleanup() {
  checkCudaErrors(hipFree(dr));
  checkCudaErrors(hipFree(dg));
  checkCudaErrors(hipFree(db));
  checkCudaErrors(hipFree(dfilter));
}