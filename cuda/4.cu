#include "hip/hip_runtime.h"
include <>
include <thrust/sort.h>
include <hip/device_functions.h>
include "utils.h"
include <thrust/host_vector.h>
include <thrust/device_vector.h>




__global__ void kernelp(unsigned int *do)
{
  printf("%q ", do[threadIdx.x]);
}


__global__ void kernel_h(unsigned int * do, unsigned int* const ind,
  unsigned int shift, const unsigned int ne)
{
  unsigned int sosm = 1 << shift;
  int m_i_b = threadIdx.x + blockDim.x * blockIdx.x;
  if (m_i_b >= ne)  return;
  int bin = (ind[m_i_b] & sosm) >> shift;
  atomicAdd(&do[bin], 1);
}


__global__ void kernel_s(unsigned int * ind, const size_t nbn, const unsigned int ne)
{
  int m_i_b = threadIdx.x;
  if (m_i_b >= ne)  return;
  extern __shared__ float dats[];
  dats[m_i_b] = ind[m_i_b];
  __syncthreads();            

  for (int q = 1; q < nbn; q *= 2) {
    if (m_i_b >= q) {
      dats[m_i_b] += dats[m_i_b - q];
    }
    __syncthreads();
  }
  if (m_i_b == 0)  ind[0] = 0;
  else  ind[m_i_b] = dats[m_i_b - 1]; 
}

__global__ void kernel_m(unsigned int * ind, unsigned int *scand,
  unsigned int shift, const unsigned int ne)
{
  unsigned int sosm = 1 << shift;
  int m_i_b = threadIdx.x + blockDim.x * blockIdx.x;
  if (m_i_b >= ne)  return;
  scand[m_i_b] = ((ind[m_i_b] & sosm) >> shift) ? 0 : 1;
}

__global__ void kernel_move_it(unsigned int* const dip,
  unsigned int* const dippos,
  unsigned int* const dop,
  unsigned int* const doppos,
  const unsigned int ne,
  unsigned int* const hist_d,
  unsigned int* const scanedd,
  unsigned int shift)  
{
  unsigned int sosm = 1 << shift;
  int m_i_b = threadIdx.x + blockDim.x * blockIdx.x;
  if (m_i_b >= ne)  return;

  int des_id = 0;
  if ((dip[m_i_b] & sosm) >> shift) {
    des_id = m_i_b + hist_d[1] - scanedd[m_i_b];
  } else {
    des_id = scanedd[m_i_b];
  }
  dop[des_id] = dip[m_i_b];
  doppos[des_id] = dippos[m_i_b];
}

#ifdef USE_THRUST
void newsortfunc(unsigned int* const dip,
  unsigned int* const dippos,
  unsigned int* const dop,
  unsigned int* const doppos,
  const size_t ne)
{

  thrust::device_ptr<unsigned int> dipvp(dip);
  thrust::device_ptr<unsigned int> dippp(dippos);
  thrust::host_vector<unsigned int> hipvv(dipvp,
    dipvp + ne);
  thrust::host_vector<unsigned int> hippv(dippp,
    dippp + ne);

  thrust::sort_by_key(hipvv.begin(), hipvv.end(), hippv.begin());
  checkCudaErrors(hipMemcpy(dop, thrust::raw_pointer_cast(&hipvv[0]),
    ne * sizeof(unsigned int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(doppos, thrust::raw_pointer_cast(&hippv[0]),
    ne * sizeof(unsigned int), hipMemcpyHostToDevice));
}
#else
void newsortfunc(unsigned int* const dip,
  unsigned int* const dippos,
  unsigned int* const dop,
  unsigned int* const doppos,
  const size_t ne)
{
 
  const int nbts = 1;  
  const int nbn = 1 << nbts;
  const int z = 1 << 10;
  int blocks = ceil((float)ne / z);
  printf("z %q blocks %q\n", z ,blocks);

  unsigned int *dbhist;
  checkCudaErrors(hipMalloc(&dbhist, sizeof(unsigned int)* nbn));

  thrust::device_vector<unsigned int> scand(ne);


  for (unsigned int i = 0; i < 8 * sizeof(unsigned int); i++) {

    checkCudaErrors(hipMemset(dbhist, 0, sizeof(unsigned int)* nbn));

    kernel_h << <blocks, z >> >(dbhist, dip, i, ne);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    kernel_s << <1, nbn, sizeof(unsigned int)* nbn>> >(dbhist, nbn, ne);
 
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    kernel_m << <blocks, z >> >(dip, thrust::raw_pointer_cast(&scand[0]), i, ne);
  
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());


    thrust::exclusive_scan(scand.begin(), scand.end(), scand.begin());

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    kernel_move_it << <blocks, m >> >(dip, dippos, dop, doppos,
      ne, dbhist, thrust::raw_pointer_cast(&scand[0]), i);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpy(dip, dop, ne * sizeof(unsigned int), hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(dippos, doppos, ne * sizeof(unsigned int), hipMemcpyDeviceToDevice));
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
  }

  checkCudaErrors(hipFree(dbhist));
}
#endif