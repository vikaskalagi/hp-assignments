#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__ void rgba_to_greyscale(const uchar4* const rgbimg,unsigned char* const greyscaleimg,int nr, int nc)
{
  
      
      int xindex = threadIdx.x;  
      int yindex = threadIdx.y;
      
      int bx = blockIdx.x;
      int by = blockIdx.y;

      int xdimg = gridDim.x;
      int ydimg = gridDim.y;

      int xdimb = blockDim.x;
      int ydimb = blockDim.y; 
      
      
      int posx = xdimb * bx + xindex;
      int posy = ydimb * by + yindex;
          
      int off1d =  posy * (xdimb * xdimg) + posx;
      
      uchar4 rgba = rgbimg[off1d];  
      float csum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
      greyscaleimg[off1d] = csum; 
    
}

void convert_to_gs(const uchar4 * const h_rgbaImage, uchar4 * const rbgimg_d,
                            unsigned char* const greyscale_d, size_t nr, size_t nc)
{
  
  const dim3 bs(nr/16+1, nc/16+1, 1); 

  const dim3 gs( 16, 16, 1);  
  
  rgba_to_greyscale<<<gs, bs>>>(rbgimg_d, greyscale_d, nr, nc);
  
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());
}